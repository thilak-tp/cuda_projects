// A Simple CUDA Program that prints Hello world after processing it in the GPU instead of the CPU
#include<hip/hip_runtime.h>  //Add CUDA library
#include<stdio.h> //For printf function

//kernel to print Hello World!
__global__ void HelloFromGPU()
{
  printf("Hello World from GPU!\n");
}


int main(){
  
  // Launch the kernel
  HelloFromGPU<<<1, 1>>>(); 
  // Wait for GPU to finish before continuing to CPU
  hipDeviceSynchronize();
  return 0;
}   