
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<iostream>

using std::cout;
using std::cin;
using std::endl;

// Kernel to Add two numbers
__global__ void Add(int *a, int *b, int *c)
{
  *c = *a + *b;
}  

//Main function starts here
int main(void) {
  int a, b, c; // host copies of a, b, c
  int *d_a, *d_b, *d_c; // device copies of a, b, c
  int size = sizeof(int);
  
  // Allocate space for device copies of a, b, c
  hipMalloc((void **)&d_a, size);
  hipMalloc((void **)&d_b, size);
  hipMalloc((void **)&d_c, size);
  
  cout<<"Enter the two numbers to add:"<<endl;
  cin>>a>>b;
  
  // Addition on the Device: main()
  // Copy inputs to device
  hipMemcpy(d_a, &a, size, hipMemcpyHostToDevice);
  hipMemcpy(d_b, &b, size, hipMemcpyHostToDevice);
  
  // Launch add() kernel on GPU
  Add<<<1,1>>>(d_a, d_b, d_c);
  
  // Copy result back to host
  hipMemcpy(&c, d_c, size, hipMemcpyDeviceToHost);
  
  // Cleanup
  hipFree(d_a); hipFree(d_b); hipFree(d_c);
  
  cout<<c;
  return 0;
}