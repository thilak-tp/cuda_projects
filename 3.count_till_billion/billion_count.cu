#include<hip/hip_runtime.h>
#include<time.h>
#include<iostream>
#include<stdio.h>

// Using only standard couts and endls
using std::cout;
using std::endl;


// GPU kernel to count from 1 to a billion
__global__ void GPUCountBillion(int *i)
{
  
  for(*i=0; (*i) < 1000000000; (*i)++);
   // printf("%d\n",*i);

}

// CPU function to count from 1 to a billion
void CPUCountBillion()
{
  for(int i =0; i < 1000000000; i++);
    //cout<<i<<endl;
    
}

// Main function 
int main()
{
  //Allocation GPU memoery for i interator
  int *i;
  hipMalloc((void **)&i,sizeof(int));
  
  //Before CPU function call timestamp
  clock_t tic = clock();
 
  //CPU function call
  CPUCountBillion();

  //After CPU function call timestamp
  clock_t toc = clock();
  //Difference between after and before gives the total seconds taken for the CPU function to be executed
  double cpu_time = double (toc - tic ) / CLOCKS_PER_SEC;
  
  //Before GPU Kernel call timestamp
  clock_t tic1 = clock();

  //GPU kernel call with 1 block each of a single thread
  GPUCountBillion<<<1,1>>>(i);
  
  //After GPU Kernel call timestamp
  clock_t toc1 = clock();
  //Difference between after and before gives the total seconds taken for the GPU kernel to be executed
  double gpu_time = double (toc1 - tic1 ) / CLOCKS_PER_SEC;

  cout<<"CPU Time for Execution "<<cpu_time<<endl;
  cout<<"GPU Time for Execution "<<gpu_time;

}