#include<hip/hip_runtime.h>
#include<iostream>
#include<time.h>
using std::cout;
using std::endl;

//Function to run on CPU for vector addition
void CPUVectorAdd(float *h_vector_a, float *h_vector_b, float *h_sum_ab, long long *n)
{
  for(int i=0;i<*n;i++)
  {
    h_sum_ab[i] = h_vector_a[i] + h_vector_b[i];
  }

}

//Kernel to run on GPU for vector addition
__global__ void GPUVectorAdd(float *d_vector_a, float *d_vector_b, float *d_sum_ab, long long *n)
{
  long long id = blockIdx.x*blockDim.x+threadIdx.x;
  if(id < *n)
    d_sum_ab[id] = d_vector_a[id] + d_vector_b[id];
}


int main()
{

  // Number of elements in the arrays. A and B.
  long long n = 1000000000;
  float *h_vector_a,*h_vector_b, *h_sum_ab, *h_gpu_op;
  float *d_vector_a,*d_vector_b, *d_sum_ab;
  long long *N;
  
  //Calcutate the size of the vectors in terms of space required
  long long size = n * sizeof(float);

  clock_t tic2  = clock();
  //Allocation of Host (CPU) memory for vectors
  h_vector_a = (float*)malloc(size);
  h_vector_b = (float*)malloc(size);
  h_sum_ab = (float*)malloc(size);
  h_gpu_op = (float*)malloc(size);
  clock_t toc2 = clock();
  float host_dyn_mem_time = float(toc2 - tic2)/CLOCKS_PER_SEC;
  cout<<"Time of Host dynamic allocation of memory: "<<host_dyn_mem_time<<endl;
  //cout<<"Reached before cuda malloc";
  //Allocation of Device (GPU) memory for vectors
  
  clock_t tic3  = clock();
  hipMalloc(&d_vector_a,size);
  hipMalloc(&d_vector_b,size);
  hipMalloc(&d_sum_ab,size);
  hipMalloc(&N,sizeof(long long));
  clock_t toc3 = clock();
  float device_dyn_mem_time = float(toc3 - tic3)/CLOCKS_PER_SEC;
  cout<<"Time of Device dynamic allocation of memory: "<<device_dyn_mem_time<<endl;

  clock_t tic4  = clock();
  for(int i = 0;i < n; i++)
  {
    h_vector_a[i] = 1.5;
    h_vector_b[i] = 5.5;
    h_sum_ab[i] = 0;
    h_gpu_op[i] = 0;
  }
  clock_t toc4 = clock();
  float host_init_time = float(toc4 - tic4)/CLOCKS_PER_SEC;
  cout<<"Time of for host initialization of vectors: "<<host_init_time<<endl;

  clock_t tic = clock();
  CPUVectorAdd(h_vector_a, h_vector_b, h_sum_ab,&n);
  clock_t toc = clock();
  float cpu_time = float(toc - tic) / CLOCKS_PER_SEC;
  cout<<"CPU Time for execution: "<<cpu_time<<endl;
  //for(int i=0 ;i <n;i++)
  // cout<<h_sum_ab[i]<<endl;



  //cout<<"Reached after CPU time execution";
  //Copy the vectors A and B into the GPU memories
  clock_t tic5 = clock();
  hipMemcpy(d_vector_a,h_vector_a,size, hipMemcpyHostToDevice);
  hipMemcpy(d_vector_a,h_vector_a,size, hipMemcpyHostToDevice);
  hipMemcpy(d_sum_ab,h_sum_ab,size, hipMemcpyHostToDevice);
  hipMemcpy(N,&n,sizeof(long long),hipMemcpyHostToDevice);
  clock_t toc5 = clock();
  float host_to_device_cpy = float(toc5 - tic5)/CLOCKS_PER_SEC;
  cout<<"Host to device Copy time: "<<host_to_device_cpy<<endl;
  
  int n_Threads = 1024;
  int n_blocks = (float)n/n_Threads;
  clock_t tic6 = clock();
  hipMemcpy(h_gpu_op,d_sum_ab,size,hipMemcpyDeviceToHost);
  clock_t toc6 = clock();
  float device_to_host_cpy = float(toc6 - tic6)/CLOCKS_PER_SEC;
  cout<<"Device to host Copy time: "<<device_to_host_cpy<<endl;

  //cout<<"Reached after op copy";
  clock_t tic1 = clock();
  GPUVectorAdd<<<n_blocks,n_Threads>>>(d_vector_a, d_vector_b, d_sum_ab, N);
  clock_t toc1 = clock();
  float gpu_time = float(toc1 - tic1) / CLOCKS_PER_SEC;
  cout<<"GPU Time for execution: "<<gpu_time<<endl;
  // for(int i=0;i<n;i++)
  //   cout<<h_gpu_op[i]<<endl;

  free(h_gpu_op);
  free(h_sum_ab);
  free(h_vector_a);
  free(h_vector_b);
  hipFree(d_sum_ab);
  hipFree(d_vector_a);
  hipFree(d_vector_b);
  hipFree(N);
  
  return 0;
}

