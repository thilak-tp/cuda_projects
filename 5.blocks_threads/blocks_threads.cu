#include<hip/hip_runtime.h>
#include<iostream>
// To use the hardware resources efficiently, we have to give the grid and thread count in a way we map the data to the threads.
using std::cout;
using std::endl;

int main()
{
  // dim3 are CUDA 3D vectors which have x,y and z coordinates. These are used to access particular threads inside particular blocks.
  //By defaults dim3 are initialized with 1,1,1 by the compiler.
  dim3 n_threads, n_blocks;
  cout<<"Threads Default: "<<n_threads.x<<", "<<n_threads.y<<", "<<n_threads.z<<", "<<" Blocks Default: "<<n_blocks.x<<", "<<n_blocks.y<<", "<<n_blocks.z<<", "<<endl;
  // We here having given a grid with 1 X 2 X 3 = 6 blocks and each block will have 3 X 4 X 5  = 60 threads so in total we'll have:
  // Total threads = no. of threads per block * no. of blocks in grid = 6 * 60 = 360 Threads
  n_blocks = {1,2,3};
  n_threads = {3,4,5};
  cout<<"Threads: "<<n_threads.x<<", "<<n_threads.y<<", "<<n_threads.z<<", "<<" Blocks: "<<n_blocks.x<<", "<<n_blocks.y<<", "<<n_blocks.z<<", "<<endl;
  //The number of threads in a block times the number of blocks gives the total number of threads that the kernel is executed in
  long total_threads = ( n_threads.x * n_threads.y * n_threads.z) * (n_blocks.x * n_blocks.y * n_blocks.z);
  cout<<"Total Threads: "<<total_threads;
  
  //The above n_threads and n_blocks can be passed to the kernel call to specify with hows many threads and blocks , that kernel will run on the GPU
  return 0;
}